#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "async_cuda.h"


#define CUDA_CALL(X) { hipError_t err = X; if (err != hipSuccess) { throw std::runtime_error(hipGetErrorString(err)); } hipError_t err_last = hipGetLastError(); if (err_last != hipSuccess) { throw std::runtime_error(hipGetErrorString(err_last)); } }

#define CUDADRIVER_CALL(func)                          \
  { hipError_t err;                                     \
    err = func;                                       \
    if (hipSuccess != err) {                        \
      char buf[100] = {0,}; \
      snprintf(buf, 100, "CUDA runtime API error: %d", err); \
      throw std::runtime_error(buf);              \
    }                                                 \
  }

namespace sys {

namespace cuda {

static hipStream_t stream_main = 0, stream_workload = 0;
static hipEvent_t event = 0;
static bool initialized = false;

void d2h_transfer(char*, char*, unsigned long) {
    assert(0 && "not implemented");
}

void h2d_transfer(char*, char*, unsigned long) {
    assert(0 && "not implemented");
}

void init_contexts()
{
    CUDA_CALL(hipStreamCreateWithFlags(&stream_main, hipStreamNonBlocking))
    CUDA_CALL(hipStreamCreateWithFlags(&stream_workload, hipStreamNonBlocking))
    CUDA_CALL(hipEventCreateWithFlags(&event, hipEventDisableTiming));
}

void sync_context()
{
    CUDA_CALL(hipStreamSynchronize(stream_main));
    CUDA_CALL(hipStreamSynchronize(stream_workload));
}

size_t get_num_of_devices() {
    int n;
    CUDA_CALL(hipGetDeviceCount(&n));
    return n;
}

void set_current_device(unsigned long n) {
    assert(!initialized);
    std::cout << "GPU device set: cuda_id=" << n << std::endl;
    CUDA_CALL(hipSetDevice(n));
    init_contexts();
    initialized = true;
    assert(0 && "not implemented");
}

void set_current(const std::string &pci_id)
{
    assert(!initialized);
    hipDevice_t dev;
    char devname[256];
    CUDADRIVER_CALL(hipInit(0));
    CUDADRIVER_CALL(hipDeviceGetByPCIBusId(&dev, pci_id.c_str()));
    CUDADRIVER_CALL(hipDeviceGetName(devname, 256, dev));
    std::cout << "GPU device set: pci_id=" << pci_id << ", name=" << devname << " (with hwloc)" << std::endl;
    initialized = true;
}

bool is_idle()
{
    if (event) {
        CUDA_CALL(hipEventRecord(event, stream_workload));
        hipError_t ret = hipEventQuery(event);
        if (ret != hipErrorNotReady && ret != hipSuccess) {
            // error case: throw exception
            CUDA_CALL(ret);
        }
        if (ret == hipErrorNotReady) {
            // stream has some load currently, not idle
            return false;
        }
    }
    return true;
}

template <int SIZE>
__global__ void workload(int ncycles, int CALIBRATION_CONST) {
    __shared__ double a[SIZE][SIZE], b[SIZE][SIZE], c[SIZE][SIZE];
    while (ncycles--) {
        for (int N = 0; N < CALIBRATION_CONST; N++) {
            for (int i = 0; i < SIZE; i++) {
                for (int j = 0; j < SIZE; j++) {
                    for (int k = 0; k < SIZE; k++) {
                        c[i][j] += a[i][k] * b[k][j] + N * N;
                    }
                }
            }
        }
    }
}

void submit_workload(int ncycles, int calibration_const)
{
    constexpr int array_dim = 10;
    workload<array_dim><<<1, 1, 0, stream_workload>>>(ncycles, calibration_const);
}

void host_alloc(char*& ptr, size_t size) {
    CUDA_CALL(hipHostAlloc(&ptr, size, hipHostMallocPortable));
}

void register_mem(char* ptr, size_t size) {
    CUDA_CALL(hipHostRegister(ptr, size, hipHostRegisterPortable));    
}

void unregister_mem(char* ptr) {
    CUDA_CALL(hipHostUnregister(ptr)); 
}

void device_alloc(char*& ptr, size_t size) {
    CUDA_CALL(hipMalloc(&ptr, size));
    CUDA_CALL(hipMemset(ptr, 0, size));
}

void host_free(char *ptr) {
    CUDA_CALL(hipHostFree(ptr));    
}

void device_free(char *ptr) {
    if (ptr) {
        CUDA_CALL(hipFree(ptr));
    }
}



}

}

