#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "async_cuda.h"
#include "async_timer.h"


#define CUDA_CALL(X) { hipError_t err = X; if (err != hipSuccess) { throw std::runtime_error(hipGetErrorString(err)); } hipError_t err_last = hipGetLastError(); if (err_last != hipSuccess) { throw std::runtime_error(hipGetErrorString(err_last)); } }

#define CUDADRIVER_CALL(func)                          \
  { hipError_t err;                                     \
    err = func;                                       \
    if (hipSuccess != err) {                        \
      char buf[100] = {0,}; \
      snprintf(buf, 100, "CUDA runtime API error: %d", err); \
      throw std::runtime_error(buf);              \
    }                                                 \
  }

namespace sys {

namespace cuda {

static hipStream_t stream_main = 0, stream_workload = 0;
static hipEvent_t event = 0;
static bool initialized = false;

void init_contexts()
{
    CUDA_CALL(hipStreamCreateWithFlags(&stream_main, hipStreamNonBlocking))
    CUDA_CALL(hipStreamCreateWithFlags(&stream_workload, hipStreamNonBlocking))
    CUDA_CALL(hipEventCreateWithFlags(&event, hipEventDisableTiming));
}

void sync_contexts()
{
    CUDA_CALL(hipStreamSynchronize(stream_main));
    CUDA_CALL(hipStreamSynchronize(stream_workload));
}

size_t get_num_of_devices() {
    int n;
    CUDA_CALL(hipGetDeviceCount(&n));
    return n;
}

void set_current_device(unsigned long n) {
    assert(!initialized);
    std::cout << "GPU device set: cuda_id=" << n << std::endl;
    CUDA_CALL(hipSetDevice(n));
    init_contexts();
    initialized = true;
}

void set_current(const std::string &pci_id)
{
    assert(!initialized);
    hipDevice_t dev;
    char devname[256];
    CUDADRIVER_CALL(hipInit(0));
    CUDADRIVER_CALL(hipDeviceGetByPCIBusId(&dev, pci_id.c_str()));
    CUDADRIVER_CALL(hipDeviceGetName(devname, 256, dev));
    std::cout << "GPU device set: pci_id=" << pci_id << ", name=" << devname << " (with hwloc)" << std::endl;
    initialized = true;
}

int get_current_device_hash() {
    int device_id;
    char ptr[1024];
    size_t len = 1024;
    memset(ptr, 0, len);
    CUDA_CALL(hipGetDevice(&device_id));
    CUDA_CALL(hipDeviceGetPCIBusId(ptr, len, device_id));
    int hash = 0;
    for (size_t i = 0; i < len / 4; i += 4) {
        if (!ptr[i*4])
            break;
        int32_t *iptr = (int32_t *)ptr + i;
        int32_t x = *iptr;
        hash ^= x;
    }
    return hash;
}

bool is_device_idle()
{
    if (event) {
        CUDA_CALL(hipEventRecord(event, stream_workload));
        hipError_t ret = hipEventQuery(event);
        if (ret != hipErrorNotReady && ret != hipSuccess) {
            // error case: throw exception
            CUDA_CALL(ret);
        }
        if (ret == hipErrorNotReady) {
            // stream has some load currently, not idle
            return false;
        }
    }
    return true;
}

template <int SIZE>
__global__ void workload(int ncycles, int CALIBRATION_CONST) {
    __shared__ double a[SIZE][SIZE], b[SIZE][SIZE], c[SIZE][SIZE];
    while (ncycles--) {
        for (int N = 0; N < CALIBRATION_CONST; N++) {
            for (int i = 0; i < SIZE; i++) {
                for (int j = 0; j < SIZE; j++) {
                    for (int k = 0; k < SIZE; k++) {
                        c[i][j] += a[i][k] * b[k][j] + N * N * ncycles;
                    }
                }
            }
        }
    }
}

void submit_workload(int ncycles, int calibration_const)
{
    constexpr int array_dim = 8;
    workload<array_dim><<<1, 1, 0, stream_workload>>>(ncycles, calibration_const);
}

int workload_calibration() {
    // Workload execution time calibration procedure. Trying to tune number of cycles
    // so that workload execution+sync time is about 100 usec
    static int cuda_workload_calibration = -1;
    if (cuda_workload_calibration != -1)
        return cuda_workload_calibration;
    cuda_workload_calibration = 1;
    const int workload_tune_maxiter = 23;
    const long target_exec_time_in_usecs = 200L;
    const long good_enough_calibration = (long)(0.95 * target_exec_time_in_usecs);
    for (int i = 0; i < workload_tune_maxiter; i++) {
        timer t;
        sys::cuda::submit_workload(1, cuda_workload_calibration);
        sys::cuda::sync_contexts();
        long execution_time_in_usecs = (long)t.stop();
        // Skip 13 first time estimations: they often include some GPU API
        // initialization time
        std::cout << ">> CUDA: execution_time_in_usecs=" << execution_time_in_usecs << " cuda_workload_calibration=" << cuda_workload_calibration << std::endl;      if (i < 13)
            continue;

        if (execution_time_in_usecs == 0)
            break;
        if (execution_time_in_usecs < good_enough_calibration) {
            auto c = int(target_exec_time_in_usecs / execution_time_in_usecs);
            if (c == 2) {
                cuda_workload_calibration *= 1.5;
            } else if (c > 1000) {
                continue;
            } else if (c > 2) {
                cuda_workload_calibration *= (c - 1);
            } else {
                auto _5percent = (int)(cuda_workload_calibration * 0.05);
                cuda_workload_calibration += (_5percent ? _5percent : 1);
            }
        } else {
            break;
        }
    }
    if (cuda_workload_calibration < 2 || cuda_workload_calibration > 1000) {
        std::cout << ">> cuda_workload_calibration=" << cuda_workload_calibration << std::endl;
        throw std::runtime_error("cuda workload calibration failed");
    }
    return cuda_workload_calibration;
    //std::cout << ">> CUDA: cuda_workload_calibration = " << cuda_workload_calibration << std::endl;
}

void host_alloc(char*& ptr, size_t size) {
    CUDA_CALL(hipHostAlloc(&ptr, size, hipHostMallocPortable));
}

void register_mem(char* ptr, size_t size) {
    CUDA_CALL(hipHostRegister(ptr, size, hipHostRegisterPortable));    
}

void unregister_mem(char* ptr) {
    CUDA_CALL(hipHostUnregister(ptr)); 
}

void device_alloc(char*& ptr, size_t size) {
    CUDA_CALL(hipMalloc(&ptr, size));
    CUDA_CALL(hipMemset(ptr, 0, size));
}

void host_free(char *ptr) {
    CUDA_CALL(hipHostFree(ptr));    
}

void device_free(char *ptr) {
    if (ptr) {
        CUDA_CALL(hipFree(ptr));
    }
}

void d2h_transfer(char *to, char *from, size_t size, transfer_t type)
{
    CUDA_CALL(hipMemcpyAsync(to, from, size, hipMemcpyDeviceToHost,
                              type == transfer_t::MAIN ? stream_main : stream_workload));
    if (type == transfer_t::MAIN) {
        CUDA_CALL(hipStreamSynchronize(stream_main))
    }
}

void h2d_transfer(char *to, char *from, size_t size, transfer_t type)
{
    CUDA_CALL(hipMemcpyAsync(to, from, size, hipMemcpyHostToDevice,
                              type == transfer_t::MAIN ? stream_main : stream_workload));
    if (type == transfer_t::MAIN) {
        CUDA_CALL(hipStreamSynchronize(stream_main))
    }
}


}

}

